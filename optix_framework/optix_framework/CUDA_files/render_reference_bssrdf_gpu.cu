#include "hip/hip_runtime.h"
#include <device_common_data.h>
#include <color_helpers.h>
#include <ray_trace_helpers.h>
#include <environment_map.h>
#include <math_helpers.h>
#include <colormap.h>
using namespace optix;

// Window variables
rtBuffer<float4, 2> output_buffer;
rtBuffer<float, 2> resulting_flux;
rtBuffer<int, 1>  photon_counter;
rtDeclareVariable(float, reference_scale_multiplier, , );
rtDeclareVariable(unsigned int, show_false_colors, , );

RT_PROGRAM void render_ref()
{
	float2 uv = make_float2(launch_index) / make_float2(launch_dim);
	float2 ip = uv * 2 - make_float2(1); // [-1, 1], this is xd, yd

	// Inverting the projection in the paper:
	float phi_o = atan2f(ip.y, ip.x);
	float l = length(ip);
	float theta_o_normalized = l; // This is |theta_o|, but we remember it is positive.
	float phi_o_normalized = normalize_angle(phi_o) / (2.0f * M_PIf);

	// Normalizing
	float2 coords = make_float2(phi_o_normalized, theta_o_normalized);
	uint2 coords_idx = make_uint2(coords * make_float2(resulting_flux.size()));
	if (l >= 1)
		output_buffer[launch_index] = make_float4(0);
	else
	{
		if (show_false_colors == 1)
			output_buffer[launch_index] = make_float4(jet(reference_scale_multiplier * resulting_flux[coords_idx] / ((float)photon_counter[0])), 1);
		else
			output_buffer[launch_index] = make_float4(reference_scale_multiplier * resulting_flux[coords_idx] / ((float)photon_counter[0]));

	}

}
