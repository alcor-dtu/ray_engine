#include "hip/hip_runtime.h"
// 02576 OptiX Rendering Framework
// Written by Jeppe Revall Frisvad, 2011
// Copyright (c) DTU Informatics 2011

#include <device_common_data.h>
#include <color_helpers.h>
#include <device_environment_map.h>

// Standard ray variables
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow, prd_shadow, rtPayload, );

using optix::rtTex2D;

__device__ __forceinline__ void get_environment_map_color(const float3& direction, float3 & color)
{
    const float2 uv = direction_to_uv_coord_cubemap(direction, optix::Matrix3x3::identity());
    color = make_float3(rtTex2D<float4>(envmap_properties->environment_map_tex_id, uv.x, uv.y)) *envmap_properties->lightmap_multiplier;
}

// Miss program returning background color
RT_PROGRAM void miss()
{
  float3 color = make_float3(0.0f);
  if (prd_radiance.flags & RayFlags::USE_EMISSION)
  {
	get_environment_map_color(ray.direction, color);
  }
  prd_radiance.result = color;
  optix_print("Ray miss, hit envmap. Returning color %f %f %f\n", color.x, color.y, color.z);
}

// Miss program returning background color
RT_PROGRAM void miss_shadow()
{
	float cos_theta;
	float3 color = make_float3(0.0f);
	get_environment_map_color(ray.direction, color);
	prd_shadow.emission = color;
	prd_shadow.attenuation = 1.0f;
	optix_print("Shadow ray miss, hit envmap. Returning color %f %f %f\n", color.x, color.y, color.z);
}
