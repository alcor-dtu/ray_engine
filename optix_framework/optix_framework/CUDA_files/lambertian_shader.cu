#include "hip/hip_runtime.h"
// 02576 OptiX Rendering Framework
// Written by Jeppe Revall Frisvad, 2011
// Copyright (c) DTU Informatics 2011

#include <device_common_data.h>
#include <light.h>
#include <random.h>
#include <sampling_helpers.h>
#include <color_helpers.h>
#include <ray_trace_helpers.h>
#include <structs_device.h>
#include <optical_helper.h>
#include <environment_map.h>


using namespace optix;

// Standard ray variables
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow, prd_shadow, rtPayload, );
rtDeclareVariable(PerRayData_cache, prd_cache, rtPayload, );

// Variables for shading
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, texcoord, attribute texcoord, );


rtDeclareVariable(float3, ambient_light_color, , );
rtDeclareVariable(float3, W, , );
rtDeclareVariable(float, phong_exp, , );
rtDeclareVariable(uint, ray_traced_reflection, , );

// Material properties (corresponding to OBJ mtl params)
rtTextureSampler<float4, 2> ambient_map;
rtTextureSampler<float4, 2> diffuse_map;
rtTextureSampler<float4, 2> specular_map;


// Monte carlo variables
rtDeclareVariable(unsigned int, N, , );
rtDeclareVariable(int, max_depth, , );
rtDeclareVariable(int, max_splits, , );
rtDeclareVariable(int, use_split, , );
rtDeclareVariable(uint, frame, , );


rtDeclareVariable(float3, eye, , );

// Any hit program for shadows
RT_PROGRAM void any_hit_shadow() {
	float3 emission = make_float3(tex2D(ambient_map, texcoord.x, texcoord.y));
	 //optix_print("%f %f %f", emission.x,emission.y,emission.z);
	shadow_hit(prd_shadow, emission);
}

__inline__ __device__ float3 sample_procedural_tex(float3 & position_local) 
{
	const float3 dims = make_float3(19.0f, 1.3f, 24.7f) * 0.5;
	const float3 black = make_float3(0.1f);
	const float3 white = make_float3(1.0f);

	if (position_local.y < dims.y - 0.01)
		return black;
	if (position_local.x < -dims.x + 1.0f || position_local.x > dims.x - 1.0f || position_local.z < -dims.z + 1.0f || position_local.z > dims.z - 1.0f)
		return black;

	if (position_local.x > -dims.x + 2.0f && position_local.x < dims.x - 2.0f && position_local.z > -dims.z + 2.3f && position_local.z < dims.z - 6.1f)
	{
		position_local = position_local - dims + make_float3(2.0f, 2.3f, 0.0f);
		float3 col = white;
		bool r = (int)(position_local.x / 1.5) % 2 == 0;
		bool c = (int)(position_local.z / 1.5) % 2 == 0;
		if (r && c || !r && !c)
		{
			col = black;
		}

		return col;
	}
	return white;
}


__inline__ __device__ float3 get_k_d()
{
	float3 k_d = make_float3(tex2D(diffuse_map, texcoord.x, texcoord.y));
	//float3 k_d = make_float3(texcoord.x, texcoord.y, 0);
	return k_d;
}

__inline__ __device__ float3 shade_specular(const float3& hit_pos, const float3 & normal, const float3 & light_vector, const float3& light_radiance, const float3 & view)
{
	const float3 k_d = get_k_d();
	float3 color = light_radiance * k_d * M_1_PIf * max(dot(normal, light_vector), 0.0f);
	// Specular
	//const float3 k_s = make_float3(tex2D(specular_map, texcoord.x, texcoord.y));
	//const float shininess = phong_exp;
	//const float3 half_vector = -normalize(light_vector + view);
	//float ndoth = pow(max(0.0f, dot(normal, half_vector)), shininess);
	//color += light_radiance * k_s * ndoth;
	return color;
}

// Closest hit program for Lambertian shading using the basic light as a directional source + specular term (blinn phong)
RT_PROGRAM void shade()
{
	float3 normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	float3 ffnormal = faceforward(normal, -ray.direction, normal);
	float3 k_a = make_float3(tex2D(ambient_map, texcoord.x, texcoord.y));
	float3 hit_pos = ray.origin + t_hit * ray.direction;

	hit_pos = rtTransformPoint(RT_OBJECT_TO_WORLD, hit_pos);
	prd_radiance.flags &= ~(RayFlags::USE_EMISSION); //Unset use emission

	float3 color = make_float3(0.0f);
	color += k_a;  
	//optix_print("%f", k_a.x);
	float3 view = normalize(W);
	uint s = prd_radiance.seed;
	for (int i = 0; i < light_size(); ++i)
	{
		// Diffuse
		
		HitInfo data(hit_pos, normal);
		for (unsigned int i = 0; i < light_size(); i++)
		{
			float3 direct = make_float3(0);
			int M = 20;
			for (int j = 0; j < M; j++)
			{
				float3 light_vector;
				float3 light_radiance;
				int cast_shadows;
				s = lcg(s);
				evaluate_direct_light(data.hit_point, data.hit_normal, light_vector, light_radiance, cast_shadows, s, i);
				float attenuation = 1.0f;
				direct += shade_specular(hit_pos, ffnormal, light_vector, light_radiance, view);
			}
			color += direct / static_cast<float>(M);
		}
	}
	prd_radiance.result = color;
	prd_radiance.seed = s;
}



RT_PROGRAM void shade_path_tracing()
{
	PerRayData_radiance& radiance = (ray.ray_type == dummy_ray_type) ? prd_cache.radiance : prd_radiance;
	optix_print("Lambertian Hit\n");
	float3 k_d = get_k_d();
   float3 normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
   //float3 ffnormal = faceforward(normal, -ray.direction, normal);
   float3 hit_pos = ray.origin + t_hit * ray.direction;

	if (radiance.depth < max_depth)
	{
		uint t = radiance.seed;
		const HitInfo data(hit_pos, normal);
		// Direct illumination
		float3 direct = make_float3(0.0f); 
		for (unsigned int i = 0; i < light_size(); i++)
		{
			for (int j = 0; j < N; j++)
			{
				float3 wi, L; int sh;
				evaluate_direct_light(hit_pos, data.hit_normal, wi, L, sh, t, i);
				direct += L;
			}
		}
		direct /= static_cast<float>(N);

		float3 env = make_float3(0.0f);
		for (int j = 0; j < N; j++)
		{
			float3 wi, L; //int sh;
			//evaluate_environment_light(wi, L, sh, data, t);
			sample_environment(wi, L, data, t);
			float cos_theta = dot(wi, normal);
			if (cos_theta <= 0.0) continue;
			env += L * cos_theta;
		}
		env /= static_cast<float>(N);

		float3 emission = make_float3(0.0f);
		if (radiance.flags & RayFlags::USE_EMISSION)
		{
			// Only the first hit uses emission
			radiance.flags &= ~(RayFlags::USE_EMISSION); //Unset use emission
			emission += make_float3(tex2D(ambient_map, texcoord.x, texcoord.y));
			//if (radiance.depth > 0 && emission.x > 0)
			//	optix_print("Emission requested. Path depth %d. Emission %f %f %f", radiance.depth, emission.x, emission.y, emission.z);
		}

		// Indirect illumination
		float prob = dot(k_d, make_float3(0.33333f));
		radiance.flags |= RayFlags::HIT_DIFFUSE_SURFACE;
		float3 indirect = make_float3(0.0f);
		float random = rnd(t);
	    if(random < prob)
		{
			float xi1 = rnd(t);
			float xi2 = rnd(t);
			float3 hemi_vec = sample_hemisphere_cosine(make_float2(xi1, xi2), normal);
			PerRayData_radiance prd;
			prd.depth = radiance.depth + 1;
			prd.flags = radiance.flags;
			prd.seed = t;
			prd.colorband = radiance.colorband;

			optix::Ray ray = optix::make_Ray(hit_pos, hemi_vec, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);

			rtTrace(top_object, ray, prd);
			indirect = prd.result / prob * M_PIf; // Cosine cancels out
			radiance.seed = prd.seed;
			radiance.colorband = prd.colorband;

    }
    else
      radiance.seed = t;
	
		optix_print("Lambertian (Bounce: %d) Env: %f %f %f, Dir: %f %f %f, Ind: %f %f %f\n", radiance.depth, env.x, env.y, env.z, direct.x, direct.y, direct.z, indirect.x, indirect.y, indirect.z);
	radiance.result = emission + k_d * M_1_PIf * (env + indirect + direct);
	}
	else
	{
	  radiance.result = make_float3(0.0f);
	}

}