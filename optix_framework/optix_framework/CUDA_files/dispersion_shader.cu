#include "hip/hip_runtime.h"
// 02576 OptiX Rendering Framework
// Written by Jeppe Revall Frisvad, 2011
// Copyright (c) DTU Informatics 2011

#include <device_common_data.h>
#include <color_helpers.h>
#include <environment_map.h>
#include <material.h>
using namespace optix;

//#define USE_SPECTRAL_RENDERING

// Standard ray variables
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow, prd_shadow, rtPayload, );

// Variables for shading
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(float3, texcoord, attribute texcoord, );

rtDeclareVariable(float3, ior_complex_real_sq, , );
rtDeclareVariable(float3, ior_complex_imag_sq, , );

rtDeclareVariable(MaterialDataCommon, material, , );

// Russian roulette variables
rtDeclareVariable(int, max_splits, , );

rtBuffer<float3, 1> normalized_cie_rgb; 
rtBuffer<float, 1> normalized_cie_rgb_cdf;
rtDeclareVariable(float, normalized_cie_rgb_step, , );
rtDeclareVariable(float, normalized_cie_rgb_wavelength, , );

rtBuffer<float, 1> ior_real_spectrum;
rtDeclareVariable(float, ior_real_wavelength, , );
rtDeclareVariable(float, ior_real_step, , );

// Any hit program for shadows
RT_PROGRAM void any_hit_shadow() {
	float3 emission = make_float3(rtTex2D<float4>(material.ambient_map, texcoord.x, texcoord.y));

	shadow_hit(prd_shadow, emission);
}

__forceinline__ __device__ unsigned int cdf_bsearch(float xi)
{
	uint table_size = normalized_cie_rgb_cdf.size();
	uint middle = table_size = table_size >> 1;
	uint odd = 0;
	while (table_size > 0)
	{
		odd = table_size & 1;
		table_size = table_size >> 1;
		unsigned int tmp = table_size + odd;
		middle = xi > normalized_cie_rgb_cdf[middle] ? middle + tmp : (xi < normalized_cie_rgb_cdf[middle - 1] ? middle - tmp : middle);
	}
	return middle;
}

// Closest hit program for Lambertian shading using the basic light as a directional source + specular term (blinn phong)
RT_PROGRAM void shade()
{
	float3 color = make_float3(0.0f);


	if (prd_radiance.depth < max_depth)
	{
		float3 normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
		float3 ffnormal = faceforward(normal, -ray.direction, normal);
		float3 hit_pos = ray.origin + t_hit * ray.direction;


		PerRayData_radiance prd_refract;
		prd_refract.depth = prd_radiance.depth + 1;
		prd_refract.flags = prd_radiance.flags | RayFlags::USE_EMISSION;
		prd_refract.colorband = prd_radiance.colorband;

		PerRayData_radiance prd_refl;
		prd_refl.depth = prd_radiance.depth + 1;
		prd_refl.flags = prd_radiance.flags | RayFlags::USE_EMISSION;

		prd_refl.colorband = prd_radiance.colorband;

		Ray reflected_ray, refracted_ray;
		float R, cos_theta;
		get_glass_rays(ray, material.ior, hit_pos, normal, reflected_ray, refracted_ray, R, cos_theta);

		rtTrace(top_object, reflected_ray, prd_refl);
		color += R * prd_refl.result;
		rtTrace(top_object, refracted_ray, prd_refract);
		color += (1 - R) * prd_refract.result;

	}
	prd_radiance.result = color;
}

__device__ __forceinline__ float& get_band(optix::float3 & v, int band)
{
	return *(&v.x + band);
}

RT_PROGRAM void shade_path_tracing(void)
{

	float3 color = make_float3(0.0f);

	if (prd_radiance.depth < max_depth)
	{
		float3 normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
		float3 hit_pos = ray.origin + t_hit*ray.direction;
		hit_pos = rtTransformPoint(RT_OBJECT_TO_WORLD, hit_pos);
		uint t = prd_radiance.seed;

		PerRayData_radiance prd_new_ray;
		prd_new_ray.depth = prd_radiance.depth + 1;
		prd_new_ray.flags = prd_radiance.flags | RayFlags::USE_EMISSION;

		float3 spectral_color = make_float3(0.0f);
		float w = 0.0f;

#ifdef USE_SPECTRAL_RENDERING
		int band = 0;
		float3 c_m;

		if (prd_radiance.colorband == -1)
		{
			// Sampling a wavelenght using CIE color matching functions CDF
			float rand = rnd(t);
			unsigned int c = cdf_bsearch(rand);
			float lambda = normalized_cie_rgb_wavelength + c * normalized_cie_rgb_step;
			band = clamp( (int)floor((lambda - ior_real_wavelength) / ior_real_step), 0, ior_real_spectrum.size() -1);
			float invpdf = c > 0
				? 1.0f / (normalized_cie_rgb_cdf[c] - normalized_cie_rgb_cdf[c - 1])
				: 1.0f / normalized_cie_rgb_cdf[c];
			c_m = normalized_cie_rgb[c] * invpdf;
		}
		else
		{
			// continue using the same frequency.
			band = prd_radiance.colorband;
			c_m = make_float3(1);
		}

		float index_of_refraction = ior_real_spectrum[band];
#else
		int band = 0;
		
		if (prd_radiance.colorband == -1)
		{
			// Selecting a random color channel.
			band = int(rnd(t) * 3.0f);
			w = 3.0f;
		}
		else
		{
			// Continue using the same colorband.
			band = prd_radiance.colorband;
			w = 1.0f;
		}

		float index_of_refraction = sqrt(get_band(ior_complex_real_sq, band));
#endif
		// Setting up payload and glass rays.
		Ray reflected_ray, refracted_ray;
		float R, cos_theta;
		get_glass_rays(ray, index_of_refraction, hit_pos, normal, reflected_ray, refracted_ray, R, cos_theta);
		prd_new_ray.depth = prd_radiance.depth + 1;
		prd_new_ray.flags = prd_radiance.flags | RayFlags::USE_EMISSION;
		prd_new_ray.colorband = band;

		// Glass russian roulette
		float xi = rnd(t);
		prd_new_ray.seed = t;
		if (xi < R)
		{
			rtTrace(top_object, reflected_ray, prd_new_ray);
			spectral_color = prd_new_ray.result;
		}
		else
		{
			rtTrace(top_object, refracted_ray, prd_new_ray);
			spectral_color = prd_new_ray.result;
		}
		

		// Combine the final result.
#ifdef USE_SPECTRAL_RENDERING
		color += c_m * spectral_color;
#else
		float component = get_band(spectral_color, band);
		get_band(color, band) = component;
#endif
		prd_radiance.seed = prd_new_ray.seed;
	}


	prd_radiance.result = color;
}
