#include "hip/hip_runtime.h"
// 02576 OptiX Rendering Framework
// Written by Jeppe Revall Frisvad, 2011
// Copyright (c) DTU Informatics 2011

#include <device_common.h>
#include <color_utils.h>
#include <environment_map.h>
#include "material_device.h"

using namespace optix;

// Standard ray variables
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );

// Variables for shading
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(float2, texcoord, attribute texcoord, );

// Any hit program for shadows
RT_PROGRAM void any_hit_shadow() { rtTerminateRay(); }

// Closest hit program for drawing shading normals
RT_PROGRAM void shade()
{
    const int material_size = material_buffer.size();
    const int material_index = get_material_index(texcoord);
    prd_radiance.result = hsv2rgb((float)material_index / material_size, 1.0, 1.0);
}