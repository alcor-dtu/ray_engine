#include "hip/hip_runtime.h"
// 02576 OptiX Rendering Framework
// Written by Jeppe Revall Frisvad, 2011
// Copyright (c) DTU Informatics 2011

#include <device_common.h>
#include <color_utils.h>
#include <environment_map.h>

using namespace optix;

// Standard ray variables
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );

// Variables for shading
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(float2, texcoord, attribute texcoord, );

// Any hit program for shadows
RT_PROGRAM void any_hit_shadow() { rtTerminateRay(); }

// Closest hit program for drawing shading normals
RT_PROGRAM void shade()
{
  prd_radiance.result = make_float3(texcoord, 0);
}