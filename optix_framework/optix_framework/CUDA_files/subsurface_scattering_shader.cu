#include "hip/hip_runtime.h"
// 02576 OptiX Rendering Framework
// Written by Jeppe Revall Frisvad, 2011
// Copyright (c) DTU Informatics 2011

#define DIRPOLE
#define TRANSMIT
#define REFLECT 

#include <device_common_data.h>
#include "../math_helpers.h"
#include "../random.h"
#include "../directional_dipole.h"
#include "../optical_helper.h"
#include "../structs.h"
#include <ray_trace_helpers.h>
#include <scattering_properties.h>
#include <material.h>

using namespace optix;

//#define REFLECT

// Standard ray variables
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow,   prd_shadow,   rtPayload, );

// SS properties
rtDeclareVariable(MaterialDataCommon, material, , );

// Variables for shading
rtBuffer<PositionSample> sampling_output_buffer;
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(float3, texcoord, attribute texcoord, ); 

//rtDeclareVariable(unsigned int, bssrdf_enabled, , );

// Any hit program for shadows
RT_PROGRAM void any_hit_shadow()
{
  // this material is opaque, so it fully attenuates all shadow rays
  prd_shadow.attenuation = 0.0f;
  rtTerminateRay();
}

// Closest hit program for Lambertian shading using the basic light as a directional source
RT_PROGRAM void shade() 
{
    if (prd_radiance.depth > max_depth)
    {
        prd_radiance.result = make_float3(0.0f);
        return;
    }

    float3 n = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
    float3 xo = ray.origin + t_hit*ray.direction;
    float3 wo = -ray.direction;
    float3 no = faceforward(n, wo, n);
    ScatteringMaterialProperties& props = material.scattering_properties;
    float recip_ior = 1.0f / props.relative_ior;
    uint& t = prd_radiance.seed;
    float reflect_xi = rnd(t);
    prd_radiance.result = make_float3(0.0f);

#ifdef TRANSMIT
    float3 beam_T = make_float3(1.0f);
    float cos_theta_o = dot(wo, n);
    bool inside = cos_theta_o < 0.0f;
    if (inside)
    {
#ifdef DIRPOLE
        beam_T = expf(-t_hit*props.deltaEddExtinction);
#else
        beam_T = expf(-t_hit*props.extinction);
#endif
        float prob = (beam_T.x + beam_T.y + beam_T.z) / 3.0f;
        if (rnd(t) >= prob) return;
        beam_T /= prob;
        recip_ior = props.relative_ior;
        cos_theta_o = -cos_theta_o;
    }
    float sin_theta_t_sqr = recip_ior*recip_ior*(1.0f - cos_theta_o*cos_theta_o);
    float cos_theta_t = 1.0f;
    float R = 1.0f;
    if (sin_theta_t_sqr < 1.0f)
    {
        cos_theta_t = sqrtf(1.0f - sin_theta_t_sqr);
        R = fresnel_R(cos_theta_o, cos_theta_t, recip_ior);
    }
    if (reflect_xi >= R)
    {
        float3 wt = recip_ior*(cos_theta_o*no - wo) - no*cos_theta_t;
        PerRayData_radiance prd_refracted;
        prd_refracted.depth = prd_radiance.depth + 1;
        Ray refracted(xo, wt, radiance_ray_type, scene_epsilon);
        rtTrace(top_object, refracted, prd_refracted);
        prd_radiance.result += prd_refracted.result*beam_T;

        if (!inside)
        {
#else
    float cos_theta_o = dot(wo, no);
    float R = fresnel_R(cos_theta_o, recip_ior);
#endif

    float chosen_transport_rr = props.mean_transport;
    float3 accumulate = make_float3(0.0f);
    uint N = sampling_output_buffer.size();

    for (uint i = 0; i < N; ++i)
    {
        PositionSample& sample = sampling_output_buffer[i];

        // compute direction of the transmitted light
        const float3& wi = sample.dir;
        float cos_theta_i = max(dot(wi, sample.normal), 0.0f);
        float cos_theta_i_sqr = cos_theta_i*cos_theta_i;
        float sin_theta_t_sqr = recip_ior*recip_ior*(1.0f - cos_theta_i_sqr);
        float cos_theta_t = sqrt(1.0f - sin_theta_t_sqr);
        float3 w12 = recip_ior*(cos_theta_i*sample.normal - wi) - sample.normal*cos_theta_t;
        float T12 = 1.0f - fresnel_R(cos_theta_i, cos_theta_t, recip_ior);

        // compute contribution if sample is non-zero
        if (dot(sample.L, sample.L) > 0.0f)
        {
            // Russian roulette
            float dist = length(xo - sample.pos);
            float exp_term = exp(-dist * chosen_transport_rr);
            if (rnd(t) < exp_term)
            {
#ifdef DIRPOLE
                accumulate += T12*sample.L*bssrdf(sample.pos, sample.normal, w12, xo, no, props) / exp_term;
#else
                accumulate += T12*sample.L*bssrdf(dist, props) / exp_term;
#endif
            }
        }
    }
#ifdef TRANSMIT
    prd_radiance.result += accumulate*props.global_coeff / (float)N;
        }
    }
#else
    float T21 = 1.0f - R;
    prd_radiance.result += T21*accumulate*props.global_coeff / (float)N;
#endif
#ifdef REFLECT
    // Trace reflected ray
    if (reflect_xi < R)
    {
        float3 wr = 2.0f*cos_theta_o*no - wo;
        PerRayData_radiance prd_reflected;
        prd_reflected.depth = prd_radiance.depth + 1;
        Ray reflected(xo, wr, radiance_ray_type, scene_epsilon);
        rtTrace(top_object, reflected, prd_reflected);
        prd_radiance.result += prd_reflected.result;
    }
#endif
}