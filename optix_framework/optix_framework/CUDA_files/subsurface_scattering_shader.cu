#include "hip/hip_runtime.h"
// 02576 OptiX Rendering Framework
// Written by Jeppe Revall Frisvad, 2011
// Copyright (c) DTU Informatics 2011

#define TRANSMIT
#define REFLECT 

#include <device_common_data.h>
#include <math_helpers.h>
#include <random.h>
#include <bssrdf.h>

#include <optical_helper.h>
#include <structs.h>
#include <ray_trace_helpers.h>
#include <scattering_properties.h>
#include <material_device.h>

using namespace optix;

//#define REFLECT

// Standard ray variables
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow,   prd_shadow,   rtPayload, );

// SS properties

// Variables for shading
rtDeclareVariable(BufPtr<PositionSample>, sampling_output_buffer, , );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(float3, texcoord, attribute texcoord, ); 

//rtDeclareVariable(unsigned int, bssrdf_enabled, , );

// Any hit program for shadows
RT_PROGRAM void any_hit_shadow()
{
  // this material is opaque, so it fully attenuates all shadow rays
  prd_shadow.attenuation = 0.0f;
  rtTerminateRay();
}  

// Closest hit program for Lambertian shading using the basic light as a directional source
RT_PROGRAM void shade() 
{
    if (prd_radiance.depth > max_depth)
    {
        prd_radiance.result = make_float3(0.0f);
        return;
    }

    float3 n = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
    float3 xo = ray.origin + t_hit*ray.direction;
    float3 wo = -ray.direction;
    float3 no = faceforward(n, wo, n);
    const MaterialDataCommon & material = get_material(xo);
    const ScatteringMaterialProperties& props = material.scattering_properties;
    float recip_ior = 1.0f / material.relative_ior;
    float reflect_xi = prd_radiance.sampler->next1D();
    prd_radiance.result = make_float3(0.0f);

#ifdef TRANSMIT
    float3 beam_T = make_float3(1.0f);
    float cos_theta_o = dot(wo, n);
    bool inside = cos_theta_o < 0.0f;
    if (inside)
    {
		beam_T = get_beam_transmittance(t_hit, props);
        float prob = (beam_T.x + beam_T.y + beam_T.z) / 3.0f;
        if (prd_radiance.sampler->next1D() >= prob) return;
        beam_T /= prob;
        recip_ior = material.relative_ior;
        cos_theta_o = -cos_theta_o;
    }

	float3 wt;
	float R;
	refract(wo, n, recip_ior, wt, R);

    if (reflect_xi >= R)
    {
		PerRayData_radiance prd_refracted = prepare_new_pt_payload(prd_radiance);
		 
		Ray refracted(xo, wt,  RayType::RADIANCE, scene_epsilon);
		rtTrace(top_object, refracted, prd_refracted);
		prd_radiance.result += prd_refracted.result*beam_T;

        if (!inside)
        {
#else
    float cos_theta_o = dot(wo, no);
    float R = fresnel_R(cos_theta_o, recip_ior);
#endif

    float chosen_transport_rr = get_sampling_mfp(props);
    float3 accumulate = make_float3(0.0f);
    uint N = sampling_output_buffer.size();

    for (uint i = 0; i < N; ++i)
    {
        PositionSample& sample = sampling_output_buffer[i];

        // compute direction of the transmitted light
        const float3& wi = sample.dir;

#ifdef TEST_SAMPLING
		accumulate += make_float3(TEST_SAMPLING_W)*sample.L;
#else
        // compute contribution if sample is non-zero
        if (dot(sample.L, sample.L) > 0.0f)
        {
            // Russian roulette
            float dist = length(xo - sample.pos);
            float exp_term = exp(-dist * chosen_transport_rr);
            if (prd_radiance.sampler->next1D() < exp_term)
            {
                 BSSRDFGeometry geometry;
                 geometry.xi = sample.pos;
                 geometry.ni = sample.normal;
                 geometry.wi = wi;
                 geometry.xo = xo;
                 geometry.no = no;
                 geometry.wo = wo;
                 accumulate += sample.L*bssrdf(geometry, recip_ior, material, BSSRDFFlags::EXCLUDE_OUTGOING_FRESNEL, *prd_radiance.sampler) / exp_term;
            }
        }
#endif
    }
#ifdef TRANSMIT
    prd_radiance.result += accumulate / (float)N;
        }
    }
#else
    float T21 = 1.0f - R;
    prd_radiance.result += T21*accumulate / (float)N;
#endif
#ifdef REFLECT
    // Trace reflected ray
    if (reflect_xi < R)
    {
		float3 wr = -reflect(wo, no);
        PerRayData_radiance prd_reflected = prepare_new_pt_payload(prd_radiance);
		Ray reflected(xo, wr,  RayType::RADIANCE, scene_epsilon);
        rtTrace(top_object, reflected, prd_reflected);	
        prd_radiance.result += prd_reflected.result;
    }
#endif
}
