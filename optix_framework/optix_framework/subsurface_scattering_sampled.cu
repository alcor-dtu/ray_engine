#include "hip/hip_runtime.h"
// 02576 OptiX Rendering Framework
// Written by Jeppe Revall Frisvad, 2011
// Copyright (c) DTU Informatics 2011

#define DIRPOLE
#define TRANSMIT
#define REFLECT 

#include <device_common_data.h>
#include <math_helpers.h>
#include <random.h>
#include <directional_dipole.h>
#include <optical_helper.h>
#include <structs.h>
#include <ray_trace_helpers.h>
#include <scattering_properties.h>
#include <material_device.h>
#include <light.h>
#include <sampling_helpers.h>
#include <camera.h>
#include <device_environment_map.h>

using namespace optix;

//#define REFLECT

// Standard ray variables
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow, prd_shadow, rtPayload, );

// SS properties

rtDeclareVariable(CameraData, camera_data, , );

// Variables for shading
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(float3, texcoord, attribute texcoord, );

//rtDeclareVariable(unsigned int, bssrdf_enabled, , );

// Any hit program for shadows
RT_PROGRAM void any_hit_shadow()
{
	// this material is opaque, so it fully attenuates all shadow rays
	prd_shadow.attenuation = 0.0f;
	rtTerminateRay();
}

// Closest hit program for Lambertian shading using the basic light as a directional source
RT_PROGRAM void shade()
{
	if (prd_radiance.depth > max_depth)
	{
		prd_radiance.result = make_float3(0.0f);
		return;
	}

	float3 n = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	float3 xo = ray.origin + t_hit*ray.direction;
	float3 wo = -ray.direction;
	float3 no = faceforward(n, wo, n);
	const MaterialDataCommon & material = get_material(xo);
	const ScatteringMaterialProperties& props = material.scattering_properties;
	float recip_ior = 1.0f / props.relative_ior;
	uint& t = prd_radiance.seed;
	float reflect_xi = rnd(t);
	prd_radiance.result = make_float3(0.0f);

#ifdef TRANSMIT
	float3 beam_T = make_float3(1.0f);
	float cos_theta_o = dot(wo, n);
	bool inside = cos_theta_o < 0.0f;
	if (inside)
	{
#ifdef DIRPOLE
		beam_T = expf(-t_hit*props.deltaEddExtinction);
#else
		beam_T = expf(-t_hit*props.extinction);
#endif
		float prob = (beam_T.x + beam_T.y + beam_T.z) / 3.0f;
		if (rnd(t) >= prob) return;
		beam_T /= prob;
		recip_ior = props.relative_ior;
		cos_theta_o = -cos_theta_o;
	}
	float sin_theta_t_sqr = recip_ior*recip_ior*(1.0f - cos_theta_o*cos_theta_o);
	float cos_theta_t = 1.0f;
	float R = 1.0f;
	if (sin_theta_t_sqr < 1.0f)
	{
		cos_theta_t = sqrtf(1.0f - sin_theta_t_sqr);
		R = fresnel_R(cos_theta_o, cos_theta_t, recip_ior);
	}
	if (reflect_xi >= R)
	{
		float3 wt = recip_ior*(cos_theta_o*no - wo) - no*cos_theta_t;
		PerRayData_radiance prd_refracted;
		prd_refracted.depth = prd_radiance.depth + 1;
		Ray refracted(xo, wt, RAY_TYPE_RADIANCE, scene_epsilon);
		rtTrace(top_object, refracted, prd_refracted);
		prd_radiance.result += prd_refracted.result*beam_T;

		if (!inside)
		{
#else
	float cos_theta_o = dot(wo, no);
	float R = fresnel_R(cos_theta_o, recip_ior);
#endif

	float chosen_transport_rr = props.mean_transport;
	float3 accumulate = make_float3(0.0f);
	uint N = 5;// sampling_output_buffer.size();

	PerRayData_normal_depth attribute_fetch_ray_payload = { make_float3(0.0f), 0.0f };
	optix::Ray attribute_fetch_ray;
	attribute_fetch_ray.ray_type = RAY_TYPE_ATTRIBUTE;
	attribute_fetch_ray.tmin = scene_epsilon;
	attribute_fetch_ray.tmax = RT_DEFAULT_MAX;
	attribute_fetch_ray.origin = camera_data.eye;


	for (uint i = 0; i < N; ++i)
	{
		optix::float2 sample = make_float2(rnd(t), rnd(t));
		float r, phi;
		optix::float2 disc_sample = sample_disk_exponential(sample, chosen_transport_rr, r, phi);
		optix::float3 to, bo;
		create_onb(no, to, bo);
		optix::float3 sample_on_tangent_plane = xo + to*disc_sample.x + bo*disc_sample.y;
		attribute_fetch_ray.direction = normalize(sample_on_tangent_plane - camera_data.eye);

		rtTrace(top_object, attribute_fetch_ray, attribute_fetch_ray_payload);

		// Real hit point
		optix::float3 xi = attribute_fetch_ray.origin + attribute_fetch_ray_payload.depth * attribute_fetch_ray.direction;
		optix::float3 ni = attribute_fetch_ray_payload.normal;
		optix::float3 wi = make_float3(0);
		optix::float3 Li;
		sample_environment(wi, Li, HitInfo(xi, ni), t);

		// compute direction of the transmitted light
		float cos_theta_i = max(dot(wi, ni), 0.0f);
		float cos_theta_i_sqr = cos_theta_i*cos_theta_i;
		float sin_theta_t_sqr = recip_ior*recip_ior*(1.0f - cos_theta_i_sqr);
		float cos_theta_t = sqrt(1.0f - sin_theta_t_sqr);
		float3 w12 = recip_ior*(cos_theta_i*ni - wi) - ni*cos_theta_t;
		float T12 = 1.0f - fresnel_R(cos_theta_i, cos_theta_t, recip_ior);

		// compute contribution if sample is non-zero
		if (dot(Li, Li) > 0.0f)
		{
			float3 S = bssrdf(xi, ni, w12, xo, no, props) * props.global_coeff;
			float dist = length(xo - xi);
			float exp_term = exp(-dist * chosen_transport_rr);
			float pdf = chosen_transport_rr * exp_term / (2.0f* M_PIf);
			accumulate += M_PIf * Li * S * T12 * cos_theta_i * r / pdf;

			// Russian roulette
//			float dist = length(xo - xi);
//			float exp_term = exp(-dist * chosen_transport_rr);
//			float r = rnd(t);
//			optix_print("T %f %f\n", exp_term, dist);
//			//if (r < exp_term)
//			{
//#ifdef DIRPOLE
//				accumulate += T12*Li*bssrdf(xi, ni, w12, xo, no, props) / exp_term;
//#else
//				accumulate += T12*sample.L*bssrdf(dist, props) / exp_term;
//#endif
//		}
		}
	}
#ifdef TRANSMIT
	prd_radiance.result += accumulate / (float)N;
		}
	}
#else
	float T21 = 1.0f - R;
	prd_radiance.result += T21*accumulate*props.global_coeff / (float)N;
#endif
#ifdef REFLECT
	// Trace reflected ray
	if (reflect_xi < R)
	{
		float3 wr = 2.0f*cos_theta_o*no - wo;
		PerRayData_radiance prd_reflected;
		prd_reflected.depth = prd_radiance.depth + 1;
		Ray reflected(xo, wr, RAY_TYPE_RADIANCE, scene_epsilon);
		rtTrace(top_object, reflected, prd_reflected);
		prd_radiance.result += prd_reflected.result;
	}
#endif
}