#include "hip/hip_runtime.h"
// 02576 OptiX Rendering Framework
// Written by Jeppe Revall Frisvad, 2011
// Copyright (c) DTU Informatics 2011

#define TRANSMIT
#define REFLECT 

#include <device_common_data.h>
#include <math_helpers.h>
#include <random.h>
#include <bssrdf.h>
#include <optical_helper.h>
#include <structs.h>
#include <ray_trace_helpers.h>
#include <scattering_properties.h>
#include <material_device.h>
#include <light.h>
#include <sampling_helpers.h>
#include <camera.h>
#include <device_environment_map.h>

using namespace optix;

//#define REFLECT

// Standard ray variables
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow, prd_shadow, rtPayload, );

// SS properties

rtDeclareVariable(CameraData, camera_data, , );

// Variables for shading
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(float3, texcoord, attribute texcoord, );

//rtDeclareVariable(unsigned int, bssrdf_enabled, , );

// Any hit program for shadows
RT_PROGRAM void any_hit_shadow()
{
	// this material is opaque, so it fully attenuates all shadow rays
	prd_shadow.attenuation = 0.0f;
	rtTerminateRay();
}

__device__ __forceinline__ bool importance_sample_position(const float3 & xo, const float3 & no, const float3 & wo, const ScatteringMaterialProperties& props, uint & t,
	float3 & xi, float3 & ni, float & integration_factor)
{
	float cos_theta_o = abs(dot(wo, no));
	float chosen_transport_rr = props.min_transport;
	PerRayData_normal_depth attribute_fetch_ray_payload = { make_float3(0.0f), RT_DEFAULT_MAX };
	optix::Ray attribute_fetch_ray;
	attribute_fetch_ray.ray_type = RAY_TYPE_ATTRIBUTE;
	attribute_fetch_ray.tmin = scene_epsilon;
	
	attribute_fetch_ray.origin = camera_data.eye;

	optix::float2 sample = make_float2(rnd(t), rnd(t));
	float r, phi;
	optix::float2 disc_sample = sample_disk_exponential(sample, chosen_transport_rr, r, phi);
	float3 sample_ray_dir;
	float3 sample_ray_origin;
	float t_max;
	optix::float3 to, bo;
	create_onb(no, to, bo);
	integration_factor = 1.0f;

	switch (bssrdf_sampling_properties->sampling_method)
	{
	case BSSRDF_SAMPLING_CAMERA_BASED_MERTENS:
	{
		float3 sample_on_tangent_plane = xo + to*disc_sample.x + bo*disc_sample.y;
		sample_ray_dir = normalize(sample_on_tangent_plane - camera_data.eye);
		sample_ray_origin = camera_data.eye;
		t_max = RT_DEFAULT_MAX;

		// Correction for camera based sampling. (see http://onlinelibrary.wiley.com/doi/10.1111/j.1467-8659.2005.00827.x/full)
		float3 d = camera_data.eye - xi;
		float3 d_prime = camera_data.eye - sample_on_tangent_plane;
		float cos_alpha_prime = dot(-sample_ray_dir, no);
		float cos_alpha = dot(-sample_ray_dir, ni);
		float jacobian = cos_alpha / cos_alpha_prime * dot(d_prime, d_prime) / dot(d, d);
		//integration_factor = abs(jacobian);
	}
		break;
	case BSSRDF_SAMPLING_NORMAL_BASED_HERY:
	{
		float3 sample_on_tangent_plane = xo + to*disc_sample.x + bo*disc_sample.y;
		sample_ray_dir = -no;
		sample_ray_origin = sample_on_tangent_plane + no * bssrdf_sampling_properties->R_max;
		t_max = bssrdf_sampling_properties->R_max * 2.0f;
	}
	break;
	case BSSRDF_SAMPLING_MIS_KING:
	{
		optix::float3 axes[3] = { no, bo, to };
		float var = rnd(t);
		int main_axis = 0;
		float* mis_weights = reinterpret_cast<float*>(&bssrdf_sampling_properties->mis_weights);

		if (var > mis_weights[0])
		{
			if (var > mis_weights[0]+ mis_weights[1])
			{
				// to on top
				main_axis = 2;
			}
			else
			{
				// bo on top
				main_axis = 1;
			}
		}

		optix_print("Axis: %d\n", main_axis);
		float3 top = axes[main_axis];
		float3 t1 = axes[(main_axis + 1) % 3];
		float3 t2 = axes[(main_axis + 2) % 3];
		float3 sample_on_tangent_plane = xo + t1*disc_sample.x + t2*disc_sample.y;
		sample_ray_origin = sample_on_tangent_plane + top * bssrdf_sampling_properties->R_max;
		sample_ray_dir = -top;
		t_max = 2.0f * bssrdf_sampling_properties->R_max;
		integration_factor /= mis_weights[main_axis];
	}
	break;
	}

	attribute_fetch_ray_payload.depth = t_max;
	attribute_fetch_ray.tmax = t_max;
	attribute_fetch_ray.direction = sample_ray_dir;
	attribute_fetch_ray.origin = sample_ray_origin; // sample_on_tangent_plane + no * 1.0f;

	rtTrace(top_object, attribute_fetch_ray, attribute_fetch_ray_payload);
	optix_print("Depth ray: %s\n", abs(attribute_fetch_ray_payload.depth - t_max) < 1e-3 ? "Miss" : "Hit");

	if (abs(attribute_fetch_ray_payload.depth - t_max) < 1e-3) // Miss
		return false;

	xi = attribute_fetch_ray.origin + attribute_fetch_ray_payload.depth * attribute_fetch_ray.direction;
	ni = attribute_fetch_ray_payload.normal;

	float dist = length(xo - xi);
	float pdf_disk = chosen_transport_rr * exp(-dist * chosen_transport_rr) / (2.0f* M_PIf);
	integration_factor *= r / pdf_disk;

	if (bssrdf_sampling_properties->sampling_method == BSSRDF_SAMPLING_CAMERA_BASED_MERTENS)
	{
		

	}
	return true;
}

// Closest hit program for Lambertian shading using the basic light as a directional source
RT_PROGRAM void shade()
{
	if (prd_radiance.depth > max_depth)
	{
		prd_radiance.result = make_float3(0.0f);
		return;
	}

	optix_print("Depth %d/%d\n", prd_radiance.depth, max_depth);

	float3 n = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	float3 xo = ray.origin + t_hit*ray.direction;
	float3 wo = -ray.direction;
	float3 no = faceforward(n, wo, n);
	const MaterialDataCommon & material = get_material(xo);
	const ScatteringMaterialProperties& props = material.scattering_properties;
	float recip_ior = 1.0f / props.relative_ior;
	uint& t = prd_radiance.seed;
	float reflect_xi = rnd(t);
	prd_radiance.result = make_float3(0.0f);

#ifdef TRANSMIT
	float3 beam_T = make_float3(1.0f);
	float cos_theta_o = dot(wo, n);
	bool inside = cos_theta_o < 0.0f;
	if (inside)
	{
		beam_T = get_beam_transmittance(t_hit, props);
		float prob = (beam_T.x + beam_T.y + beam_T.z) / 3.0f;
		if (rnd(t) >= prob) return;
		beam_T /= prob;
		recip_ior = props.relative_ior;
		cos_theta_o = -cos_theta_o;
	}

	float sin_theta_t_sqr = recip_ior*recip_ior*(1.0f - cos_theta_o*cos_theta_o);
	float cos_theta_t = 1.0f;
	float R = 1.0f;
	if (sin_theta_t_sqr < 1.0f)
	{
		cos_theta_t = sqrtf(1.0f - sin_theta_t_sqr);
		R = fresnel_R(cos_theta_o, cos_theta_t, recip_ior);
	}
	if (reflect_xi >= R)
	{
		float3 wt = recip_ior*(cos_theta_o*no - wo) - no*cos_theta_t;
		PerRayData_radiance prd_refracted;
		prd_refracted.depth = prd_radiance.depth + 1;
		Ray refracted(xo, wt, RAY_TYPE_RADIANCE, scene_epsilon);
		rtTrace(top_object, refracted, prd_refracted);
		prd_radiance.result += prd_refracted.result*beam_T;

		if (!inside)
		{
#else
	float cos_theta_o = dot(wo, no);
	float R = fresnel_R(cos_theta_o, recip_ior);
#endif

	
	float3 L_d = make_float3(0.0f);
	uint N = 1;// sampling_output_buffer.size();

	int count = 0;

	for (uint i = 0; i < N; i++)
	{
		float integration_factor;
		float3 xi, ni;
		importance_sample_position(xo, no, wo, props, t, xi, ni, integration_factor);
		// Real hit point
		
		optix::float3 wi = make_float3(0);
		optix::float3 L_i;
		sample_environment(wi, L_i, HitInfo(xi, ni), t); // This returns pre-sampled w_i and L_i

		// compute direction of the transmitted light
		float cos_theta_i = max(dot(wi, ni), 0.0f);
		float cos_theta_i_sqr = cos_theta_i*cos_theta_i;
		float sin_theta_t_sqr = recip_ior*recip_ior*(1.0f - cos_theta_i_sqr);
		float cos_theta_t = sqrt(1.0f - sin_theta_t_sqr);
		float3 w12 = recip_ior*(cos_theta_i*ni - wi) - ni*cos_theta_t;
		float T12 = 1.0f - fresnel_R(cos_theta_i, cos_theta_t, recip_ior);

		// compute contribution if sample is non-zero
		if (dot(L_i, L_i) > 0.0f)
		{
			float3 S_d = bssrdf(xi, ni, w12, xo, no, props);
			L_d += L_i * S_d * T12 * integration_factor;
			count++;
		}
	}
	count = max(1, count);
#ifdef TRANSMIT
	prd_radiance.result += L_d / (float)count;
		}
	}
#else
	float T21 = 1.0f - R;
	prd_radiance.result += T21*accumulate / (float)count;
#endif
#ifdef REFLECT
	// Trace reflected ray
	if (reflect_xi < R)
	{
		float3 wr = 2.0f*cos_theta_o*no - wo;
		PerRayData_radiance prd_reflected;
		prd_reflected.depth = prd_radiance.depth + 1;
		Ray reflected(xo, wr, RAY_TYPE_RADIANCE, scene_epsilon);
		rtTrace(top_object, reflected, prd_reflected);
		prd_radiance.result += prd_reflected.result;
	}
#endif
}