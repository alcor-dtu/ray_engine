#include "hip/hip_runtime.h"
// 02576 OptiX Rendering Framework
// Written by Jeppe Revall Frisvad, 2011
// Copyright (c) DTU Informatics 2011

#define TRANSMIT
#define REFLECT 

#include <device_common_data.h>
#include <math_helpers.h>
#include <random.h>
#include <bssrdf.h>
#include <optical_helper.h>
#include <structs.h>
#include <ray_trace_helpers.h>
#include <scattering_properties.h>
#include <material_device.h>
#include <light.h>
#include <sampling_helpers.h>
#include <camera.h>
#include <device_environment_map.h>

using namespace optix;

//#define REFLECT

// Standard ray variables
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow, prd_shadow, rtPayload, );

// SS properties

rtDeclareVariable(CameraData, camera_data, , );

// Variables for shading
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(float3, texcoord, attribute texcoord, );
rtDeclareVariable(unsigned int, samples_per_pixel, , );

rtDeclareVariable(BufPtr<BSSRDFSamplingProperties>, bssrdf_sampling_properties, , );

// Any hit program for shadows
RT_PROGRAM void any_hit_shadow()
{
	// this material is opaque, so it fully attenuates all shadow rays
	prd_shadow.attenuation = 0.0f;
	rtTerminateRay();
}

__device__ __forceinline__ bool importance_sample_position(const float3 & xo, const float3 & no, const float3 & wo, const ScatteringMaterialProperties& props, uint & t,
	float3 & xi, float3 & ni, float & integration_factor)
{
	float cos_theta_o = abs(dot(wo, no));
	float chosen_sampling_mfp = get_sampling_mfp(props);
	PerRayData_normal_depth attribute_fetch_ray_payload = { make_float3(0.0f), RT_DEFAULT_MAX };
	optix::Ray attribute_fetch_ray;
	attribute_fetch_ray.ray_type = RAY_TYPE_ATTRIBUTE;
	attribute_fetch_ray.tmin = scene_epsilon;
	
	attribute_fetch_ray.origin = camera_data.eye;

	optix::float2 sample = make_float2(rnd(t), rnd(t));
	float r, phi;
	optix::float2 disc_sample = sample_disk_exponential(sample, chosen_sampling_mfp, r, phi);
	r = max(bssrdf_sampling_properties->R_min, r);
	float3 sample_ray_dir;
	float3 sample_ray_origin;
	float t_max;
	optix::float3 to, bo;
	create_onb(no, to, bo);
	integration_factor = 1.0f;
	float3 sample_on_tangent_plane;

	switch (bssrdf_sampling_properties->sampling_method)
	{
	case BSSRDF_SAMPLING_CAMERA_BASED_MERTENS:
	{
		sample_on_tangent_plane = xo + to*disc_sample.x + bo*disc_sample.y;
		sample_ray_dir = normalize(sample_on_tangent_plane - camera_data.eye);
		sample_ray_origin = camera_data.eye;
		t_max = RT_DEFAULT_MAX;
	}
		break;
	case BSSRDF_SAMPLING_NORMAL_BASED_HERY:
	{
		sample_on_tangent_plane = xo + to*disc_sample.x + bo*disc_sample.y;
		sample_ray_dir = -no;
		sample_ray_origin = sample_on_tangent_plane + no * bssrdf_sampling_properties->d_max;
		t_max = RT_DEFAULT_MAX; 
	}
	break;
	case BSSRDF_SAMPLING_MIS_KING:
	{
		optix::float3 axes[3] = { no, bo, to };
		float var = rnd(t);
		int main_axis = 0;
		float* mis_weights = reinterpret_cast<float*>(&bssrdf_sampling_properties->mis_weights);

		if (var > mis_weights[0])
		{
			if (var > mis_weights[0]+ mis_weights[1])
			{
				// to on top
				main_axis = 2;
			}
			else
			{
				// bo on top
				main_axis = 1;
			}
		}

		float3 top = axes[main_axis];
		float3 t1 = axes[(main_axis + 1) % 3];
		float3 t2 = axes[(main_axis + 2) % 3];
		sample_on_tangent_plane = xo + t1*disc_sample.x + t2*disc_sample.y;
		sample_ray_origin = sample_on_tangent_plane + top * bssrdf_sampling_properties->d_max;
		sample_ray_dir = -top;
		t_max = RT_DEFAULT_MAX; //2.0f * bssrdf_sampling_properties->R_max;
		integration_factor /= mis_weights[main_axis];
	}
	break;
	}

	attribute_fetch_ray_payload.depth = t_max;
	attribute_fetch_ray.tmax = t_max;
	attribute_fetch_ray.direction = sample_ray_dir;
	attribute_fetch_ray.origin = sample_ray_origin; 

	rtTrace(current_geometry_node, attribute_fetch_ray, attribute_fetch_ray_payload);
//	optix_print("Depth ray: %s\n", abs(attribute_fetch_ray_payload.depth - t_max) < 1e-3 ? "Miss" : "Hit");

	if (abs(attribute_fetch_ray_payload.depth - t_max) < 1e-3) // Miss
		return false;

	xi = attribute_fetch_ray.origin + attribute_fetch_ray_payload.depth * attribute_fetch_ray.direction;
	ni = attribute_fetch_ray_payload.normal;

	float pdf_disk = chosen_sampling_mfp * exp(-r * chosen_sampling_mfp) / (2.0f* M_PIf);
	integration_factor *= r / pdf_disk;
	optix_print("r: %f, pdf_disk %f, inte %f\n", r, pdf_disk, integration_factor);

	if (bssrdf_sampling_properties->sampling_method == BSSRDF_SAMPLING_CAMERA_BASED_MERTENS
		&& bssrdf_sampling_properties->correct_camera == 1)
	{
		float3 d = camera_data.eye - xi;
		float cos_alpha = dot(-sample_ray_dir, ni);

		float3 d_tan = camera_data.eye - sample_on_tangent_plane;
		float cos_alpha_tan = dot(-sample_ray_dir, no);

		float jacobian = max(1e-3, cos_alpha_tan) / max(1e-3,cos_alpha) * max(1e-3, dot(d, d)) / max(1e-3, dot(d_tan, d_tan));
		integration_factor *= jacobian;
	}
	if (bssrdf_sampling_properties->sampling_method == BSSRDF_SAMPLING_NORMAL_BASED_HERY
		)
	{
		float inv_jac = dot(normalize(no), normalize(ni));
		optix_print("Dot no ni: %f\n", inv_jac);
		integration_factor = inv_jac > 0.0f? integration_factor/inv_jac : 0.0f;
	}
	return true;
}


// Closest hit program for Lambertian shading using the basic light as a directional source
__device__ __forceinline__ void _shade()
{
	if (prd_radiance.depth > max_depth)
	{
		prd_radiance.result = make_float3(0.0f);
		return;
	}

	float3 n = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	float3 xo = ray.origin + t_hit*ray.direction;
	float3 wo = -ray.direction;
	float3 no = faceforward(n, wo, n);
	const MaterialDataCommon & material = get_material(xo);
	const ScatteringMaterialProperties& props = material.scattering_properties;
	float recip_ior = 1.0f / material.relative_ior;
	uint& t = prd_radiance.seed;
	float reflect_xi = rnd(t);
	prd_radiance.result = make_float3(0.0f);

#ifdef TRANSMIT
	float3 beam_T = make_float3(1.0f);
	float cos_theta_o = dot(wo, n);
	bool inside = cos_theta_o < 0.0f;
	if (inside)
	{
		beam_T = get_beam_transmittance(t_hit, props);
		float prob = (beam_T.x + beam_T.y + beam_T.z) / 3.0f;
		if (rnd(t) >= prob) return;
		beam_T /= prob;
		recip_ior = material.relative_ior;
		cos_theta_o = -cos_theta_o;
	}

	float sin_theta_t_sqr = recip_ior*recip_ior*(1.0f - cos_theta_o*cos_theta_o);
	float cos_theta_t = 1.0f;
	float R = 1.0f;
	if (sin_theta_t_sqr < 1.0f)
	{
		cos_theta_t = sqrtf(1.0f - sin_theta_t_sqr);
		R = fresnel_R(cos_theta_o, cos_theta_t, recip_ior);
	}

	R = bssrdf_sampling_properties->show_mode == BSSRDF_SHADERS_SHOW_REFLECTION ? 1.0f : R;
	R = bssrdf_sampling_properties->show_mode == BSSRDF_SHADERS_SHOW_REFRACTION ? 0.0f : R;

	if (reflect_xi >= R)
	{
		float3 wt = recip_ior*(cos_theta_o*no - wo) - no*cos_theta_t;
		PerRayData_radiance prd_refracted = prepare_new_pt_payload(prd_radiance);

		Ray refracted(xo, wt, RAY_TYPE_RADIANCE, scene_epsilon);
		rtTrace(top_object, refracted, prd_refracted);

		prd_radiance.seed = prd_refracted.seed;
		prd_radiance.result += prd_refracted.result*beam_T;

		if (!inside)
		{
#else
	float cos_theta_o = dot(wo, no);
	float R = fresnel_R(cos_theta_o, recip_ior);
#endif

	float3 L_d = make_float3(0.0f);
	uint N = samples_per_pixel;// sampling_output_buffer.size();

	int count = 0;

	for (uint i = 0; i < N; i++)
	{
		float integration_factor;
		float3 xi, ni;
		if (!importance_sample_position(xo, no, wo, props, t, xi, ni, integration_factor))
			continue;
		// Real hit point
		
		optix::float3 wi = make_float3(0);
		optix::float3 L_i;
		sample_light(xi, ni, 0, t, wi, L_i); // This returns pre-sampled w_i and L_i

		// compute direction of the transmitted light
		float cos_theta_i = max(dot(wi, ni), 0.0f);
		float cos_theta_i_sqr = cos_theta_i*cos_theta_i;
		float sin_theta_t_sqr = recip_ior*recip_ior*(1.0f - cos_theta_i_sqr);
		float cos_theta_t_i = sqrt(1.0f - sin_theta_t_sqr);
		float3 w12 = recip_ior*(cos_theta_i*ni - wi) - ni*cos_theta_t_i;
		float T12 = 1.0f - fresnel_R(cos_theta_i, cos_theta_t_i, recip_ior);

		float3 w21 = no * cos_theta_t - recip_ior * (cos_theta_o * no - wo);

		// compute contribution if sample is non-zero
		if (dot(L_i, L_i) > 0.0f)
		{
			float3 S_d = bssrdf(xi, ni, w12, xo, no, w21, props);
			L_d += L_i * S_d * T12 * integration_factor;
			optix_print("Ld %f %f %f Li %f %f %f T12 %f int %f\n", L_d.x, L_d.y, L_d.z, L_i.x, L_i.y, L_i.z, T12, integration_factor);
		}
	}
#ifdef TRANSMIT
		prd_radiance.result += L_d / (float)N;
		}
	}
#else
	float T21 = 1.0f - R;
	prd_radiance.result += T21*accumulate / (float)count;
#endif
#ifdef REFLECT
	// Trace reflected ray
	if (reflect_xi < R)
	{
		float3 wr = 2.0f*cos_theta_o*no - wo;
		PerRayData_radiance prd_reflected = prepare_new_pt_payload(prd_radiance);
		Ray reflected(xo, wr, RAY_TYPE_RADIANCE, scene_epsilon);
		rtTrace(top_object, reflected, prd_reflected);

		prd_radiance.seed = prd_reflected.seed;
		prd_radiance.result += prd_reflected.result;
	}
#endif
}

RT_PROGRAM void shade() { _shade(); }
RT_PROGRAM void shade_path_tracing() { _shade(); }