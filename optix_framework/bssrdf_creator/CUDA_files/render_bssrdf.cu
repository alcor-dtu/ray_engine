#include "hip/hip_runtime.h"
#include <device_common_data.h>
#include <full_bssrdf_host_device_common.h>
#include <color_helpers.h>
#include <ray_trace_helpers.h>
#include <environment_map.h>
#include <math_helpers.h>
#include <colormap.h>
#include <photon_trace_reference_bssrdf.h>
#include <scattering_properties.h>
using namespace optix;

// Window variables
rtBuffer<float4, 2> output_buffer;
rtDeclareVariable(unsigned int, show_false_colors, , ); 
rtDeclareVariable(float, reference_scale_multiplier, , );
rtDeclareVariable(TexPtr, resulting_flux_tex, , );
rtDeclareVariable(OutputShape::Type, reference_bssrdf_output_shape, , ) = OutputShape::HEMISPHERE;
rtDeclareVariable(int, reference_bssrdf_fresnel_mode, , ) = BSSRDF_RENDER_MODE_FULL_BSSRDF;
rtDeclareVariable(float, reference_bssrdf_rel_ior, , );

__device__ __forceinline__ float convert_to_tex_coordinate(float normalized_buffer_coordinate, unsigned int size)
{
	float sz = static_cast<float>(size);
	float factor = (sz - 1.0f) / sz;
	return 0.5f / sz + normalized_buffer_coordinate * factor;
}

RT_PROGRAM void render_ref()
{
	float2 uv = make_float2(launch_index) / make_float2(launch_dim);
	float2 ip = uv * 2 - make_float2(1); // [-1, 1], this is xd, yd

	if (reference_bssrdf_output_shape == OutputShape::HEMISPHERE)
	{
		// Inverting the projection in the paper:
		float phi_o = atan2f(ip.y, ip.x);
		float l = length(ip);

		if (l >= 1) 
		{
			output_buffer[launch_index] = make_float4(0); 
		}
		else 
		{
			float theta_o = M_PIf * 0.5f * l;
			float2 coords = get_normalized_hemisphere_buffer_coordinates(phi_o,theta_o);
			optix::uint3 size = optix::rtTexSize(resulting_flux_tex);
			float2 texcoords;
			texcoords.x = convert_to_tex_coordinate(coords.x, size.x);
			texcoords.y = convert_to_tex_coordinate(coords.y, size.y);

			float cos_theta_o = cosf(theta_o);
			float S = reference_scale_multiplier * optix::rtTex2D<float4>(resulting_flux_tex, texcoords.x, texcoords.y).x;
            //float S = reference_scale_multiplier * optix::rtTex2DFetch<float4>(resulting_flux_tex, int(coords.x * size.x), int(coords.y * size.y)).x;
			float T21 = 1.0f - fresnel_R(cos_theta_o, reference_bssrdf_rel_ior);

			float val;
			switch (reference_bssrdf_fresnel_mode)
			{
			case BSSRDF_RENDER_MODE_FRESNEL_OUT_ONLY: val = T21; break;
			case BSSRDF_RENDER_MODE_REMOVE_FRESNEL: val = S / T21; break;
			default:
			case BSSRDF_RENDER_MODE_FULL_BSSRDF: val = S; break;
			}

			if (show_false_colors == 1)
				output_buffer[launch_index] = make_float4(jet(val), 1);
			else
				output_buffer[launch_index] = make_float4(val);
		}
	}
	else
	{
		float fresnel_integral = C_phi(reference_bssrdf_rel_ior) * 4 * M_PIf;
		float R21;
		optix::float3 w21; 
		const optix::float3 no = optix::make_float3(0,0,1);
		const optix::float3 wo = no;
		refract(wo, no, 1 / reference_bssrdf_rel_ior, w21, R21);
		float T21 = 1.0f - R21;

		float S = optix::rtTex2D<float4>(resulting_flux_tex, uv.x, uv.y).x;
		float S_shown = reference_scale_multiplier * fresnel_integral / T21 * S;

		float t = clamp((logf(S_shown + 1.0e-10f) / 2.30258509299f + 6.0f) / 6.0f, 0.0f, 1.0f);
		float h = clamp((1.0f - t)*2.0f, 0.0f, 0.65f);

		optix::float4 res = make_float4(S_shown);
		if (show_false_colors)
		{
			// Jet, but with FD17 Color visualization:
			float Slog = log10(S);
			Slog = (Slog + 7.0f) / 8.0f; // Between -7 and 1
			Slog = clamp(Slog, 0.0f, 1.0f);
			res = optix::make_float4(jet(Slog), 1);
			// Standard matlab Jet
			//res = optix::make_float4(jet(S_shown), 1);
			// Jeppe visualization
			res = optix::make_float4(hsv2rgb(h, 1.0, 1.0), 1.0);
		}
		output_buffer[launch_index] = res;
	}
	
}
